// This example demonstrates the use of shared per-block variables to
// implement an optimized adjacent difference algorithm.  In this example,
// a per-block __shared__ array acts as a "bandwidth multiplier" by eliminating
// redundant loads issued by neighboring threads.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>

// compute the number of lines of code each implementation requires
const unsigned int simple_implementation_begin = __LINE__;

// a simple version of adjacent_difference which issues redundant loads from off-chip global memory
__global__ void adjacent_difference_simple(int *result, int *input)
{
  // compute this thread's global index
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i > 0)
  {
    // each thread loads two elements from global memory
    int x_i = input[i];
    int x_i_minus_one = input[i-1];

    // compute the difference using values stored in registers
    result[i] = x_i - x_i_minus_one;
  }
}
const unsigned int simple_implementation_size = __LINE__ - simple_implementation_begin;


const unsigned int optimized_implementation_begin = __LINE__;

// an optimized version of adjacent_difference which eliminates redundant loads
__global__ void adjacent_difference(int *result, int *input)
{
  // a __shared__ array with one element per thread
  // the size of the array is allocated dynamically upon kernel launch
  extern __shared__ int s_data[];

  // each thread reads one element to s_data
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

  // since one array gets allocated per-block, we index the array using our
  // per-block thread index, threadIdx
  // the global array, input, is indexed as usual
  s_data[threadIdx.x] = input[i];

  // avoid race condition: ensure all loads to s_data complete before we try to read from it
  __syncthreads();

  if(threadIdx.x > 0)
  {
    // compute the difference directly from s_data
    // it is implemented in fast, on-chip memory
    result[i] = s_data[threadIdx.x] - s_data[threadIdx.x - 1];
  }
  else if(i > 0)
  {
    // handle thread block boundary
    // the first thread in a block needs data that was read by the
    // last thread of the previous block into its shared array
    // this thread can't access that array, so issue one redundant load per block
    result[i] = s_data[threadIdx.x] - input[i-1];
  }
}
const unsigned int optimized_implementation_size = __LINE__ - optimized_implementation_begin;


int main(void)
{
  // create a large workload so we can easily measure the
  // performance difference of both implementations
  const size_t block_size = 512;
  const size_t num_blocks = (1<<24) / block_size;
  const size_t n = num_blocks * block_size;

  // generate random input on the host
  std::vector<int> h_input(n);
  std::generate(h_input.begin(), h_input.end(), rand);

  // allocate storage for the device
  int *d_input = 0, *d_result = 0;
  hipMalloc((void**)&d_input, sizeof(int) * n);
  hipMalloc((void**)&d_result, sizeof(int) * n);

  // copy input to the device
  hipMemcpy(d_input, &h_input[0], sizeof(int) * n, hipMemcpyHostToDevice);

  // time the kernel launches using CUDA events
  hipEvent_t launch_begin, launch_end;
  hipEventCreate(&launch_begin);
  hipEventCreate(&launch_end);

  // to get accurate timings, launch a single "warm-up" kernel

  // dynamically allocate the __shared__ array by passing its
  // size in bytes to the 3rd parameter of the triple chevrons
  adjacent_difference_simple<<<num_blocks,block_size,block_size*sizeof(int)>>>(d_result, d_input);

  const size_t num_launches = 100;

  // time many kernel launches and take the average time
  float average_simple_time = 0;
  for(int i = 0; i < num_launches; ++i)
  {
    // record a CUDA event immediately before and after the kernel launch
    hipEventRecord(launch_begin,0);
    adjacent_difference_simple<<<num_blocks,block_size,block_size*sizeof(int)>>>(d_result, d_input);
    hipEventRecord(launch_end,0);
    hipEventSynchronize(launch_end);

    // measure the time spent in the kernel
    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);

    average_simple_time += time;
  }
  average_simple_time /= num_launches;

  // now time the optimized kernel

  // again, launch a single "warm-up" kernel
  adjacent_difference<<<num_blocks,block_size,block_size*sizeof(int)>>>(d_result, d_input);

  // time many kernel launches and take the average time
  float average_optimized_time = 0;
  for(int i = 0; i < num_launches; ++i)
  {
    // record a CUDA event immediately before and after the kernel launch
    hipEventRecord(launch_begin,0);
    adjacent_difference<<<num_blocks,block_size,block_size*sizeof(int)>>>(d_result, d_input);
    hipEventRecord(launch_end,0);
    hipEventSynchronize(launch_end);

    // measure the time spent in the kernel
    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);

    average_optimized_time += time;
  }
  average_optimized_time /= num_launches;

  // report the effective throughput of each kernel in GB/s
  // the effective throughput is measured as size of input read + size of output written divided by time
  float simple_throughput = static_cast<float>(2 * n * sizeof(int)) / (average_simple_time / 1000.0f) / 1000000000.0f;
  float optimized_throughput = static_cast<float>(2 * n * sizeof(int)) / (average_optimized_time / 1000.0f) / 1000000000.0f;

  // compute throughput per line of code to measure how productive we were
  float simple_throughput_per_sloc = simple_throughput / simple_implementation_size;
  float optimized_throughput_per_sloc = optimized_throughput / optimized_implementation_size;

  std::cout << "Work load size: " << n << std::endl;
  std::cout << "simple implementation SLOCs: " << simple_implementation_size << std::endl;
  std::cout << "Optimized implementation SLOCs: " << optimized_implementation_size << std::endl << std::endl;

  std::cout << "Throughput of simple kernel: " << simple_throughput << " GB/s" << std::endl;
  std::cout << "Throughput of optimized kernel: " << optimized_throughput << " GB/s" << std::endl;
  std::cout << "Performance improvement: " << optimized_throughput / simple_throughput << "x" << std::endl;
  std::cout << std::endl;

  std::cout << "Throughput of simple kernel per line of code: " << simple_throughput_per_sloc << " GB/s/sloc" << std::endl;
  std::cout << "Throughput of optimized kernel per line of code: " << optimized_throughput_per_sloc << " GB/s/sloc" << std::endl;
  std::cout << "Performance improvement per line of code: " << optimized_throughput_per_sloc / simple_throughput_per_sloc << "x" << std::endl;

  // destroy the CUDA events
  hipEventDestroy(launch_begin);
  hipEventDestroy(launch_end);

  // deallocate device memory
  hipFree(d_input);
  hipFree(d_result);

  return 0;
}

