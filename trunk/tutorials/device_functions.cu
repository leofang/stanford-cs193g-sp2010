// This example introduces __device__ functions, which are special functions
// which may be called from code executing on the device.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


// __device__ functions may only be called from __global__ functions or other
// __device__ functions.  Unlike __global__ functions, __device__ functions are
// not configured, and have no restriction on return type.
__device__ int get_constant(void)
{
  // just return 7
  return 7;
}

__device__ int get_block_index(void)
{
  // return the index of the current thread's block
  return blockIdx.x;
}

__device__ int get_thread_index(void)
{
  // return the index of the current thread within its block
  return threadIdx.x;
}

__device__ int get_global_index(void)
{
  // return the index of the current thread across the entire grid launch
  return blockIdx.x * blockDim.x + threadIdx.x;
}


// kernel1 returns the result of calling the __device__ function return_constant():
__global__ void kernel1(int *array)
{
  int index = get_global_index();
  array[index] = get_constant();
}


// kernel2 returns the result of calling the __device__ function return_block_index():
__global__ void kernel2(int *array)
{
  int index = get_global_index();
  array[index] = get_block_index();
}


// kernel3 returns the result of calling the __device__ function return_thread_index():
__global__ void kernel3(int *array)
{
  int index = get_global_index();
  array[index] = get_thread_index();
}


// kernel4 returns the result of calling the __device__ function return_thread_index():
__global__ void kernel4(int *array)
{
  int index = get_global_index();
  array[index] = get_global_index();
}


int main(void)
{
  int num_elements = 256;

  int num_bytes = num_elements * sizeof(int);

  int *device_array = 0;
  int *host_array = 0;

  // malloc a host array
  host_array = (int*)malloc(num_bytes);

  // cudaMalloc a device array
  hipMalloc((void**)&device_array, num_bytes);

  // if either memory allocation failed, report an error message
  if(host_array == 0 || device_array == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // choose a launch configuration
  int block_size = 128;
  int grid_size = num_elements / block_size;

  // launch each kernel and print out the results

  kernel1<<<grid_size,block_size>>>(device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel1 results:\n");
  for(int i=0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n\n");


  kernel2<<<grid_size,block_size>>>(device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel2 results:\n");
  for(int i=0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n\n");


  kernel3<<<grid_size,block_size>>>(device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel3 results:\n");
  for(int i=0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n\n");


  kernel4<<<grid_size,block_size>>>(device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel4 results:\n");
  for(int i=0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n\n");


  // deallocate memory
  free(host_array);
  hipFree(device_array);
}

