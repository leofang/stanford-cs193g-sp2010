// #include stdlib.h for malloc/free

#include <hip/hip_runtime.h>
#include <stdlib.h>

// #include stdio.h for printf
#include <stdio.h>

// nvcc automatically #includes headers needed for cudaMalloc, cudaFree, cudaMemcpy, & cudaMemset


int main(void)
{
  // create arrays of 16 elements
  int num_elements = 16;

  // compute the size of the arrays in bytes
  int num_bytes = num_elements * sizeof(int);
  
  // pointers to host & device arrays
  int *device_array = 0;
  int *host_array = 0;
  
  // malloc a host array
  host_array = (int*)malloc(num_bytes);

  // cudaMalloc a device array
  // we pass cudaMalloc a pointer to the device_array pointer
  hipMalloc((void**)&device_array, num_bytes);
  
  // if either memory allocation failed, report an error message
  if(host_array == 0 || device_array == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }
  
  // zero out the device array with cudaMemset
  hipMemset(device_array, 0, num_bytes);

  // copy the contents of the device array to the host array to inspect the result
  // use cudaMemcpyDeviceToHost to indicate the direction of the copy
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
  
  // print out the result element by element
  for(int i=0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n");
  
  // use free to free the host array
  free(host_array);

  // use cudaFree to free the device array
  hipFree( device_array );
}

