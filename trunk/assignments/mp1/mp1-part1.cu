#include "hip/hip_runtime.h"
/* This is machine problem 1, part 1, shift cypher
 *
 * The problem is to take in a string of unsigned ints and an int,
 * the shift amount, and add the number to each element of
 * the string, effectively "shifting" each element in the 
 * string.
 * SUBMISSION GUIDELINES:
 * You should copy the complete shift_cyper function from your solution
 * into a file called mp1-part1-solution-kernel.cu and submit that file.
 * The function needs to have exactly the same interface (including __global__)
 * as the empty shift_cypher function given below. 
 */


#include <stdlib.h>
#include <stdio.h>
#include <ctime>

#include "mp1-util.h"


// Repeating from the tutorial, just in case you haven't looked at it.

// "kernels" or __global__ functions are the entry points to code that executes on the GPU
// The keyword __global__ indicates to the compiler that this function is a GPU entry point.
// __global__ functions must return void, and may only be called or "launched" from code that
// executes on the CPU.

void host_shift_cypher(uint *input_array, uint *output_array, int shift_amount, int alphabet_max, int array_length)
{
  int i;
  for(i=0;i<array_length;i++)
  {
    int element = input_array[i];
    int shifted = element + shift_amount;
    if(shifted > alphabet_max || shifted < 0)
    {
      shifted = shifted % (alphabet_max + 1);
    }
    output_array[i] = (uint)shifted;
  }
}


// This kernel implements a per element shift
__global__ void shift_cypher(uint *input_array, uint *output_array, int shift_amount, int alphabet_max, int array_length)
{
  // your code here
}


int main(void)
{
  // initialize
  srand(time(NULL));
  
  // create arrays of 16M elements
  int num_elements = 1 << 24;

  
  int alphabet_max = 45647;
  
  // compute the size of the arrays in bytes
  int num_bytes = num_elements * sizeof(uint);

  // pointers to host & device arrays
  uint *host_input_array = 0;
  uint *host_output_array = 0;
  uint *host_output_checker_array = 0;
  uint *device_input_array = 0;
  uint *device_output_array = 0;
  
  event_pair timer;
  

  // malloc host arrays
  host_input_array = (uint*)malloc(num_bytes);
  host_output_array = (uint*)malloc(num_bytes);
  host_output_checker_array = (uint*)malloc(num_bytes);

  // hipMalloc device arrays
  hipMalloc((void**)&device_input_array, num_bytes);
  hipMalloc((void**)&device_output_array, num_bytes);
  
  // if either memory allocation failed, report an error message
  if(host_input_array == 0 || host_output_array == 0 || host_output_checker_array == 0 || 
	device_input_array == 0 || device_output_array == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }


  // generate random input string
  int shift_amount = rand();
  
  for(int i=0;i< num_elements;i++)
  {
    host_input_array[i] = (uint)rand(); 
  }
  
  // do copies to and from gpu once to get rid of timing weirdness
  // on first time accesses due to driver
  hipMemcpy(device_input_array, host_input_array, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(host_output_array, device_output_array, num_bytes, hipMemcpyDeviceToHost);

  start_timer(&timer);
  // copy input to GPU
  hipMemcpy(device_input_array, host_input_array, num_bytes, hipMemcpyHostToDevice);
  check_launch("copy to gpu");
  stop_timer(&timer,"copy to gpu");
  
  // choose a number of threads per block
  // we use 512 threads here
  int block_size = 512;

  int grid_size = num_elements / block_size;

  start_timer(&timer);
  // launch kernel
  shift_cypher<<<grid_size,block_size>>>(device_input_array, device_output_array, shift_amount, alphabet_max, num_elements);
  check_launch("gpu shift cypher");
  stop_timer(&timer,"gpu shift cypher");

  start_timer(&timer);
  // download and inspect the result on the host:
  hipMemcpy(host_output_array, device_output_array, num_bytes, hipMemcpyDeviceToHost);
  check_launch("copy from gpu");
  stop_timer(&timer,"copy from gpu");
  
  start_timer(&timer);
  // generate reference output
  host_shift_cypher(host_input_array, host_output_checker_array, shift_amount, alphabet_max, num_elements);
  stop_timer(&timer,"host shift cypher");
  
  // check CUDA output versus reference output
  int error = 0;
  for(int i=0;i<num_elements;i++)
  {
    if(host_output_array[i] != host_output_checker_array[i]) 
    { 
      error = 1;
    }
  }
  
  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else
  {
    printf("Worked! CUDA and reference output match. \n");
  }
 
  // deallocate memory
  free(host_input_array);
  free(host_output_array);
  free(host_output_checker_array);
  hipFree(device_input_array);
  hipFree(device_output_array);
}

