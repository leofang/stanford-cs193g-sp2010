// This is machine problem 1, part 3, page ranking
// The problem is to compute the rank of a set of webpages
// given a link graph, aka a graph where each node is a webpage,
// and each edge is a link from one page to another.
// We're going to use the Pagerank algorithm (http://en.wikipedia.org/wiki/Pagerank),
// specifically the iterative algorithm for calculating the rank of a page
// We're going to run 20 iterations of the propage step.
// Implement the corresponding host and kernel code in CUDA.
// The result of the CUDA version should end up in h_graph_nodes_A and should be equivalent
// to the output produced by the reference version.
// We'll be testing the code with randomized graph sizes, edges per node, etc., so your
// code should not hardcode any constants.



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

 
#define EPSILON 0.0001f 

void host_graph_propagate(uint *graph_indices, uint *graph_edges, float *graph_nodes_in, float *graph_nodes_out, float * inv_edges_per_node, int array_length)
{
	for(int i=0;i<array_length;i++)
	{
		float sum = 0.f; 
		for(int j=graph_indices[i];j<graph_indices[i+1];j++)
		{
			sum += graph_nodes_in[graph_edges[j]]*inv_edges_per_node[graph_edges[j]];
		}
		graph_nodes_out[i] = 0.5f/(float)array_length + 0.5f*sum;
	}
}

void host_graph_iterate(uint *graph_indices, uint *graph_edges, float *graph_nodes_A, float *graph_nodes_B, float * inv_edges_per_node, int nr_iterations, int array_length)
{
	assert((nr_iterations % 2) == 0);
	int iter=0;
	for(;iter<nr_iterations;iter+=2)
	{
		host_graph_propagate(graph_indices, graph_edges, graph_nodes_A, graph_nodes_B, inv_edges_per_node, array_length);
		host_graph_propagate(graph_indices, graph_edges, graph_nodes_B, graph_nodes_A, inv_edges_per_node, array_length);
	}
}

// your kernel code here...


int main(void)
{
  // create arrays of 2M elements
  int num_elements = 128*128*128;
  int avg_edges = 8;
  int iterations = 20;
  
  // pointers to host & device arrays
  uint *h_graph_indices = 0;
  float *h_inv_edges_per_node = 0;
  uint *h_graph_edges = 0;
  float *h_graph_nodes_A = 0;
  float *h_graph_nodes_B = 0;
  float *h_graph_nodes_checker_A = 0;
  float *h_graph_nodes_checker_B = 0;
  
  

  // malloc host array
  // index array has to be n+1 so that the last thread can 
  // still look at its neighbor for a stopping point
  h_graph_indices = (uint*)malloc((num_elements+1) * sizeof(uint));
  h_inv_edges_per_node = (float*)malloc((num_elements) * sizeof(float));
  h_graph_edges = (uint*)malloc(num_elements * avg_edges * sizeof(uint));
  h_graph_nodes_A = (float*)malloc(num_elements * sizeof(float));
  h_graph_nodes_B = (float*)malloc(num_elements * sizeof(float));
  h_graph_nodes_checker_A = (float*)malloc(num_elements * sizeof(float));
  h_graph_nodes_checker_B = (float*)malloc(num_elements * sizeof(float));
  
  // if either memory allocation failed, report an error message
  if(h_graph_indices == 0 || h_graph_edges == 0 || h_graph_nodes_A == 0 || h_graph_nodes_B == 0 || 
	 h_inv_edges_per_node == 0 || h_graph_nodes_checker_A == 0 || h_graph_nodes_checker_B == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // generate random input string
  // initialize
  srand(1);
   
  h_graph_indices[0] = 0;
  for(int i=0;i< num_elements;i++)
  {
	// FIXME: better randomization of number of edges
	int nr_edges = (i % 15) + 1;
	h_inv_edges_per_node[i] = 1.f/(float)nr_edges;
	h_graph_indices[i+1] = h_graph_indices[i] + nr_edges;
	if(h_graph_indices[i+1] >= (num_elements * avg_edges))
	{
		printf("more edges than we have space for\n");
		return 1;
	}
	for(int j=h_graph_indices[i];j<h_graph_indices[i+1];j++)
	{
		h_graph_edges[j] = rand() % num_elements;
	}
	// FIXME: better randomization of input vector
	h_graph_nodes_A[i] =  1.f/(float)num_elements;
	h_graph_nodes_checker_A[i] =  h_graph_nodes_A[i];
  }
  

  // the result of the computation on the gpu should end up in h_graph_nodes_A
      
  // generate reference output
  host_graph_iterate(h_graph_indices, h_graph_edges, h_graph_nodes_checker_A, h_graph_nodes_checker_B, h_inv_edges_per_node, iterations, num_elements);
  
  // check CUDA output versus reference output
  int error = 0;
  for(int i=0;i<num_elements;i++)
  {
	float n = h_graph_nodes_A[i];
	float c = h_graph_nodes_checker_A[i];
	if((n - c)*(n - c) > EPSILON) 
	{
		printf("%d:%.3f::",i,h_graph_nodes_A[i] - h_graph_nodes_checker_A[i]);
		error = 1;
	}
	
  }
  
  if(error)
  {
	printf("Output of CUDA version and normal version didn't match! \n");
  }
  else {
	printf("Worked! CUDA and reference output match. \n");
  }
 
  // deallocate memory
  free(h_graph_indices);
  free(h_inv_edges_per_node);
  free(h_graph_edges);
  free(h_graph_nodes_A);
  free(h_graph_nodes_B);
  free(h_graph_nodes_checker_A);
  free(h_graph_nodes_checker_B);
}

